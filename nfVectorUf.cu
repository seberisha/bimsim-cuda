#include "hip/hip_runtime.h"
#include "nearfield.h"
#include "rts/math/spherical_bessel.h"
#include "rts/math/legendre.h"
#include <stdlib.h>
#include "rts/cuda/error.h"
#include "rts/cuda/timer.h"

//Incident field for a single plane wave
__global__ void gpuVectorUfp(bsComplex* Uf, bsVector k, ptype kmag, bsPoint f, ptype A, bsRect ABCD, int uR, int vR)
{
	/*Compute the scalar focused field using Debye focusing
		k		= direction of focused light, where |k| = 2*pi/lambda
		P		= rect struct describing the field slice
		rX, rY	= resolution of the field slice
		cNAin	= inner NA of the condenser
		cNAout	= outer NA of the condenser
	*/

	//get the current coordinate in the plane slice
	int iu = blockIdx.x * blockDim.x + threadIdx.x;
	int iv = blockIdx.y * blockDim.y + threadIdx.y;

	//make sure that the thread indices are in-bounds
	if(iu >= uR || iv >= vR) return;

	//compute the index (easier access to the scalar field array)
	int i = iv*uR + iu;

	//compute the parameters for u and v
	ptype u = (ptype)iu / uR;
	ptype v = (ptype)iv / vR;

	//get the rtsPoint in world space and then the r vector
	bsPoint p = ABCD(u, v);
	bsVector r = p - f;
	//ptype d = r.len();

	ptype k_dot_r = kmag * k.dot(r);
	bsComplex d(0, k_dot_r);

	Uf[i] = exp(d) * A;

}

//Incident field for a focused point source
__global__ void gpuVectorUf(bsComplex* Uf, bsVector k, ptype kmag, bsPoint f, ptype A, bsRect ABCD, int uR, int vR, ptype cosAlpha, ptype cosBeta, int nl, ptype j_conv = 1.4)
{
    //Compute the scalar focused field using Debye focusing
	//	k		= direction of focused light, where |k| = 2*pi/lambda
	//	P		= rect struct describing the field slice
	//	rX, rY	= resolution of the field slice
	//	cNAin	= inner NA of the condenser
	//	cNAout	= outer NA of the condenser


	//get the current coordinate in the plane slice
	int iu = blockIdx.x * blockDim.x + threadIdx.x;
	int iv = blockIdx.y * blockDim.y + threadIdx.y;

	//make sure that the thread indices are in-bounds
	if(iu >= uR || iv >= vR) return;

	//compute the index (easier access to the scalar field array)
	int i = iv*uR + iu;

	//compute the parameters for u and v
	ptype u = (ptype)iu / (uR);
	ptype v = (ptype)iv / (vR);

	//get the rtsPoint in world space and then the r vector
	bsPoint p = ABCD(u, v);
	bsVector r = p - f;
	ptype d = r.len();
	if(d < EPSILON_FLOAT)
	{
        Uf[i] = A * 2 * PI * (cosAlpha - cosBeta);
        return;
    }

	//get info for the light direction and frequency
	//k = k.norm();
	r = r.norm();

	//compute the imaginary factor i^l
	bsComplex im = bsComplex(0, 1);
	bsComplex il = bsComplex(1, 0);

	//Bessel and Legendre functions are computed dynamically to save memory
	//initialize the Bessel and Legendre functions
	ptype j[2];
	ptype kd = kmag * d;
	rts::init_sbesselj<ptype>(kd, j);

	ptype P[2];
	//get the angle between k and r (light direction and position vector)
	ptype cosTheta;
	cosTheta = k.dot(r);

	//deal with the degenerate case where r == 0
	//if(isnan(cosTheta))
    //    cosTheta = 0;
	rts::init_legendre<ptype>(cosTheta, P[0], P[1]);

	//initialize legendre functions for the cassegrain angles
	ptype Palpha[3];
	//ptype cosAlpha = cos(asin(cNAin));
	rts::init_legendre<ptype>(cosAlpha, Palpha[0], Palpha[1]);
	Palpha[2] = 1;

	ptype Pbeta[3];
	//ptype cosBeta = cos(asin(cNAout));
	rts::init_legendre<ptype>(cosBeta, Pbeta[0], Pbeta[1]);
	Pbeta[2] = 1;

	//for each order l
	bsComplex sumUf(0.0, 0.0);
	ptype jl = 0.0;
	ptype Pl;
	for(int l = 0; l<=nl; l++)
	{

		if(l==0)
		{

			jl = j[0];
			Pl = P[0];
		}
		else if(l==1)
		{
			jl = j[1];
			Pl = P[1];

			//adjust the cassegrain Legendre function
			Palpha[2] = Palpha[0];
			rts::shift_legendre<ptype>(l+1, cosAlpha, Palpha[0], Palpha[1]);
			Pbeta[2] = Pbeta[0];
			rts::shift_legendre<ptype>(l+1, cosBeta, Pbeta[0], Pbeta[1]);
		}
		else
		{
			rts::shift_sbesselj<ptype>(l, kd, j);//, j_conv);
			rts::shift_legendre<ptype>(l, cosTheta, P[0], P[1]);

			jl = j[1];
			Pl = P[1];

			//adjust the cassegrain outer Legendre function
			Palpha[2] = Palpha[0];
			rts::shift_legendre<ptype>(l+1, cosAlpha, Palpha[0], Palpha[1]);
			Pbeta[2] = Pbeta[0];
			rts::shift_legendre<ptype>(l+1, cosBeta, Pbeta[0], Pbeta[1]);
		}

		sumUf += il * jl * Pl * (Palpha[1] - Palpha[2] - Pbeta[1] + Pbeta[2]);

		il *= im;
	}

	Uf[i] = sumUf * 2 * PI * A;

}


void nearfieldStruct::vectorUf()
{


    gpuStartTimer();

	//create one thread for each pixel of the field slice
	dim3 dimBlock(SQRT_BLOCK, SQRT_BLOCK);
	dim3 dimGrid((Uf.R[0] + SQRT_BLOCK -1)/SQRT_BLOCK, (Uf.R[1] + SQRT_BLOCK - 1)/SQRT_BLOCK);

	//if we are computing a plane wave, call the gpuScalarUfp function
	if(planeWave)
	{
      //  std::cout<<"Calculating vector plane wave..."<<std::endl;
		gpuVectorUfp<<<dimGrid, dimBlock>>>(Uf.x_hat, k, 2 * PI / lambda, focus, A, pos, Uf.R[0], Uf.R[1]);
	}
	//otherwise compute the condenser info and create a focused field
	else
	{
		//pre-compute the cosine of the obscuration and objective angles
		ptype cosAlpha = cos(asin(condenser[0]));
		ptype cosBeta = cos(asin(condenser[1]));
		//compute the scalar Uf field (this will be in the x_hat channel of Uf)
		gpuVectorUf<<<dimGrid, dimBlock>>>(Uf.x_hat, k, 2 * PI / lambda, focus, A, pos, Uf.R[0], Uf.R[1], cosAlpha, cosBeta, m);
	}

	t_Uf = gpuStopTimer();
}
