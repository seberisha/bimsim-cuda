#include "hip/hip_runtime.h"
#include "nearfield.h"

#include "rts/math/legendre.h"
#include "rts/cuda/error.h"
#include "rts/cuda/timer.h"

texture<float, hipTextureType2D> texJ;

__global__ void gpuScalarUfp(bsComplex* Uf, bsVector k, ptype kmag, bsPoint f, ptype A, bsRect ABCD, int uR, int vR);

__global__ void gpuScalarUfLut(bsComplex* Uf, bsRect ABCD, int uR, int vR, bsPoint f, bsVector k, ptype A, ptype cosAlpha, ptype cosBeta, int nl, ptype dmin, ptype dmax, int dR)
{
    /*This function computes the focused field for a 2D slice

    Uf      =   destination field slice
    ABCD    =   plane representing the field slice in world space
    uR, vR  =   resolution of the Uf field
    f       =   focal point of the condenser
    k       =   direction of the incident light
    A       =   amplitude of the incident field
    cosAlpha=   cosine of the solid angle subtended by the condenser obscuration
    cosBeta =   cosine of the solid angle subtended by the condenser aperature
    nl      =   number of orders used to compute the field
    dR      =   number of Bessel function values in the look-up texture

    */

    //get the current coordinate in the plane slice
	int iu = blockIdx.x * blockDim.x + threadIdx.x;
	int iv = blockIdx.y * blockDim.y + threadIdx.y;

	//make sure that the thread indices are in-bounds
	if(iu >= uR || iv >= vR) return;

	//compute the index (easier access to the scalar field array)
	int i = iv*uR + iu;

	//compute the parameters for u and v
	ptype u = (ptype)iu / (uR);
	ptype v = (ptype)iv / (vR);

	

	//get the rtsPoint in world space and then the r vector
	bsPoint p = ABCD(u, v);
	bsVector r = p - f;
	ptype d = r.len();

	if(d == 0)
	{
        Uf[i] = A * 2 * PI * (cosAlpha - cosBeta);
        return;
    }

	//get info for the light direction and frequency
	r = r.norm();

	//compute the imaginary factor i^l
	bsComplex im = bsComplex(0, 1);
	bsComplex il = bsComplex(1, 0);

	//Legendre functions are computed dynamically to save memory
	//initialize the Legendre functions

	ptype P[2];
	//get the angle between k and r (light direction and position vector)
	ptype cosTheta;
	cosTheta = k.dot(r);

	rts::init_legendre<ptype>(cosTheta, P[0], P[1]);

	//initialize legendre functions for the cassegrain angles
	ptype Palpha[3];
	rts::init_legendre<ptype>(cosAlpha, Palpha[0], Palpha[1]);
	Palpha[2] = 1;

	ptype Pbeta[3];
	rts::init_legendre<ptype>(cosBeta, Pbeta[0], Pbeta[1]);
	Pbeta[2] = 1;

	//for each order l
	bsComplex sumUf(0, 0);
	ptype jl = 0;
	ptype Pl;
	ptype di = ( (d - dmin)/(dmax - dmin) ) * (dR - 1);
	for(int l = 0; l<=nl; l++)
	{
        jl = tex2D(texJ, l + 0.5f, di + 0.5f);
		if(l==0)
			Pl = P[0];
		else if(l==1)
		{
			Pl = P[1];

			//adjust the cassegrain Legendre function
			Palpha[2] = Palpha[0];
			rts::shift_legendre<ptype>(l+1, cosAlpha, Palpha[0], Palpha[1]);
			Pbeta[2] = Pbeta[0];
			rts::shift_legendre<ptype>(l+1, cosBeta, Pbeta[0], Pbeta[1]);
		}
		else
		{
			rts::shift_legendre<ptype>(l, cosTheta, P[0], P[1]);

			Pl = P[1];

			//adjust the cassegrain outer Legendre function
			Palpha[2] = Palpha[0];
			rts::shift_legendre<ptype>(l+1, cosAlpha, Palpha[0], Palpha[1]);
			Pbeta[2] = Pbeta[0];
			rts::shift_legendre<ptype>(l+1, cosBeta, Pbeta[0], Pbeta[1]);
		}

		sumUf += il * jl * Pl * (Palpha[1] - Palpha[2] - Pbeta[1] + Pbeta[2]);
		//sumUf += jl;

		il *= im;
	}

	Uf[i] = sumUf * 2 * PI * A;
	//Uf[i] = u;
	//return;
}

void nearfieldStruct::scalarUfLut()
{
    gpuStartTimer();
	
    //calculate the minimum and maximum points in the focused field
    d_min = pos.dist(focus);
    d_max = pos.dist_max(focus);

    //allocate space for the Bessel function
    int dR = 2 * max(Uf.R[0], Uf.R[1]);
    ptype* j = NULL;
	j = (ptype*) malloc(sizeof(ptype) * dR * (m+1));

	//calculate Bessel function LUT
	calcBesselLut(j, d_min, d_max, dR);
	
    //create a CUDA array structure and specify the format description
	hipArray* arrayJ;
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
    //allocate memory
    HANDLE_ERROR(hipMallocArray(&arrayJ, &channelDesc, m+1, dR));
	
    //specify texture properties
    texJ.addressMode[0] = hipAddressModeMirror;
    texJ.addressMode[1] = hipAddressModeMirror;
    texJ.filterMode     = hipFilterModeLinear;
    texJ.normalized     = false;

    //bind the texture to the array
    HANDLE_ERROR(hipBindTextureToArray(texJ, arrayJ, channelDesc));

    //copy the CPU Bessel LUT to the GPU-based array
    HANDLE_ERROR( hipMemcpy2DToArray(arrayJ, 0, 0, j, (m+1)*sizeof(float), (m+1)*sizeof(float), dR, hipMemcpyHostToDevice));

    //----------------Compute the focused field
    //create one thread for each pixel of the field slice
	dim3 dimBlock(SQRT_BLOCK, SQRT_BLOCK);
	dim3 dimGrid((Uf.R[0] + SQRT_BLOCK -1)/SQRT_BLOCK, (Uf.R[1] + SQRT_BLOCK - 1)/SQRT_BLOCK);

	//if we are computing a plane wave, call the gpuScalarUfp function
	if(planeWave)
	{
		gpuScalarUfp<<<dimGrid, dimBlock>>>(Uf.x_hat, k, 2 * PI / lambda, focus, A, pos, Uf.R[0], Uf.R[1]);
	}
	//otherwise compute the condenser info and create a focused field
	else
	{
		//pre-compute the cosine of the obscuration and objective angles
		ptype cosAlpha = cos(asin(condenser[0]));
		ptype cosBeta = cos(asin(condenser[1]));
		//compute the scalar Uf field (this will be in the x_hat channel of Uf)
		gpuScalarUfLut<<<dimGrid, dimBlock>>>(Uf.x_hat, pos, Uf.R[0], Uf.R[1], focus, k, A, cosAlpha, cosBeta, m, d_min, d_max, dR);
	}

	
    //free everything
	free(j);
	
	HANDLE_ERROR(hipFreeArray(arrayJ));

	t_Uf = gpuStopTimer();
}
