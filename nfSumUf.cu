#include "hip/hip_runtime.h"
#include "nearfield.h"
#include <stdlib.h>
#include "rts/cuda/error.h"
#include "rts/cuda/timer.h"

__global__ void gpuScalarUsp(bsComplex* Ufx, bsComplex* Ufy, bsComplex* Ufz,
							 bsComplex* Ux, bsComplex* Uy, bsComplex* Uz,
							 bsPoint* ps, ptype* as, int ns, bsRect ABCD, int uR, int vR)
{

	//get the current coordinate in the plane slice
	int iu = blockIdx.x * blockDim.x + threadIdx.x;
	int iv = blockIdx.y * blockDim.y + threadIdx.y;

	//make sure that the thread indices are in-bounds
	if(iu >= uR || iv >= vR) return;

	//compute the index (easier access to the scalar field array)
	int i = iv*uR + iu;

	//compute the parameters for u and v
	ptype u = (ptype)iu / uR;
	ptype v = (ptype)iv / vR;

	//get the rtsPoint in world space and then the r vector
	bsPoint p = ABCD(u, v);
	bsVector r;
	ptype d;

	//if we are inside of a sphere, return
	for(int is=0; is<ns; is++)
	{
		r = p - ps[is];
		d = r.len();
		if(d < as[is])
		{
			//printf("\t\t sumUf inside sphere -- d: %f\n",d);
			return;
		}
	}

	//otherwise add the focused field to the full field
	if(Ufx != NULL)
		Ux[i] += Ufx[i];
	if(Ufy != NULL)
		Uy[i] += Ufy[i];
	if(Ufz != NULL)
		Uz[i] += Ufz[i];
}

void nearfieldStruct::sumUf()
{


	//create arrays to store sphere positions and radii
	int nSpheres = sVector.size();

	//if the number of spheres is zero, just copy the incident field
	if(nSpheres == 0)
	{
		if(U.x_hat != NULL)
			HANDLE_ERROR(hipMemcpy(U.x_hat, Uf.x_hat, sizeof(bsComplex) * U.R[0] * U.R[1], hipMemcpyDeviceToDevice));
		if(U.y_hat != NULL)
			HANDLE_ERROR(hipMemcpy(U.y_hat, Uf.y_hat, sizeof(bsComplex) * U.R[0] * U.R[1], hipMemcpyDeviceToDevice));
		if(U.z_hat != NULL)
			HANDLE_ERROR(hipMemcpy(U.z_hat, Uf.z_hat, sizeof(bsComplex) * U.R[0] * U.R[1], hipMemcpyDeviceToDevice));
		return;
	}

	//time the calculation of the focused field
	//gpuStartTimer();

	bsPoint* cpu_p = (bsPoint*)malloc(sizeof(bsPoint) * nSpheres);
	ptype* cpu_a = (ptype*)malloc(sizeof(ptype) * nSpheres);

	//copy the sphere positions and radii to the new arrays
	for(int s=0; s<nSpheres; s++)
	{
		cpu_p[s] = sVector[s].p;
		cpu_a[s] = sVector[s].a;
	}

	//copy the arrays to the gpu
	bsPoint* gpu_p;
	HANDLE_ERROR(hipMalloc( (void**) &gpu_p, sizeof(bsPoint) * nSpheres));
	HANDLE_ERROR(hipMemcpy(gpu_p, cpu_p, sizeof(bsPoint) * nSpheres, hipMemcpyHostToDevice));
	ptype* gpu_a;
	HANDLE_ERROR(hipMalloc( (void**) &gpu_a, sizeof(ptype) * nSpheres));
	HANDLE_ERROR(hipMemcpy(gpu_a, cpu_a, sizeof(ptype) * nSpheres, hipMemcpyHostToDevice));


	//create one thread for each pixel of the field slice
	dim3 dimBlock(SQRT_BLOCK, SQRT_BLOCK);
	dim3 dimGrid((U.R[0] + SQRT_BLOCK -1)/SQRT_BLOCK, (U.R[1] + SQRT_BLOCK - 1)/SQRT_BLOCK);

	//copy the focused field
	gpuScalarUsp<<<dimGrid, dimBlock>>>(Uf.x_hat,
										Uf.y_hat,
										Uf.z_hat,
										U.x_hat,
										U.y_hat,
										U.z_hat,
										gpu_p,
										gpu_a,
										nSpheres,
										pos,
										U.R[0],
										U.R[1]);



    //free sphere lists
    HANDLE_ERROR(hipFree(gpu_p));
    HANDLE_ERROR(hipFree(gpu_a));


}
