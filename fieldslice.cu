#include "hip/hip_runtime.h"
#include "fieldslice.h"
#include "dataTypes.h"
#include "rts/cuda/error.h"
#include "rts/cuda/threads.h"

__global__ void field_intensity(bsComplex* x, bsComplex* y, bsComplex* z, ptype* I, unsigned int N)
{
    //compute the index for this thread
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ThreadIndex1D();

	if(i >= N) return;

	ptype xm = x[i].abs();


	if(y != NULL && z != NULL)
	{
		ptype ym = y[i].abs();
		ptype zm = z[i].abs();
		I[i] = xm*xm + ym*ym + zm*zm;
	}
	else
	{
		I[i] = xm*xm;
	}
}


__global__ void resample_intensity_without_integrating(bsComplex* x, bsComplex* y, bsComplex* z, ptype* D, int uR, int vR, int ss)
{
    //get the current coordinate in the plane slice
    int iu = blockIdx.x * blockDim.x + threadIdx.x;
    int iv = blockIdx.y * blockDim.y + threadIdx.y;

    //make sure that the thread indices are in-bounds
    if(iu >= uR || iv >= vR) return;

    //compute the index into the detector
    int i = iv*uR + iu;

    //compute the index into the field
    int fi;

    //initialize the intensity for the pixel to zero
    ptype I = 0;
    ptype xm = 0;
    ptype ym = 0;
    ptype zm = 0;

    int ix, iy;
    for(ix = 0; ix<ss; ix++)
        for(iy = 0; iy<ss; iy++)
        {
            //fi = iv*ss*ss*uR + iy*ss*uR + iu*ss + ix;
            fi = (iv*ss + iy)*ss*uR + iu*ss + ix;
            if(x !=NULL)
                xm = x[fi].abs();
            if(y != NULL)
                ym = y[fi].abs();
            if(z != NULL)
                zm = z[fi].abs();
            I += xm*xm + ym*ym + zm*zm;
        }

    D[i] = I/(ss*ss);
   // if (i<128)
}


__global__ void resample_intensity(bsComplex* x, bsComplex* y, bsComplex* z, ptype* D, int uR, int vR, int ss, float scale)
{
	//get the current coordinate in the plane slice
	int iu = blockIdx.x * blockDim.x + threadIdx.x;
	int iv = blockIdx.y * blockDim.y + threadIdx.y;

	//make sure that the thread indices are in-bounds
	if(iu >= uR || iv >= vR) return;

	//compute the index into the detector
	int i = iv*uR + iu;

	//compute the index into the field
	int fi;

	//initialize the intensity for the pixel to zero
	ptype I = 0;
	ptype xm = 0;
	ptype ym = 0;
	ptype zm = 0;

	int ix, iy;
	for(ix = 0; ix<ss; ix++)
		for(iy = 0; iy<ss; iy++)
		{
			//fi = iv*ss*ss*uR + iy*ss*uR + iu*ss + ix;
			fi = (iv*ss + iy)*ss*uR + iu*ss + ix;
			if(x !=NULL)
				xm = x[fi].abs();
			if(y != NULL)
				ym = y[fi].abs();
			if(z != NULL)
				zm = z[fi].abs();
			I += xm*xm + ym*ym + zm*zm;
		}

    D[i] += scale*I/(ss*ss);
}

__global__ void field_real(bsComplex* field_component, ptype* V, unsigned int N)
{
    //compute the index for this thread
	int i = ThreadIndex1D();
	if(i >= N) return;

	V[i] = field_component[i].real();
}

__global__ void field_imaginary(bsComplex* field_component, ptype* V, unsigned int N)
{
    //compute the index for this thread
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ThreadIndex1D();
	if(i >= N) return;

	V[i] = field_component[i].imag();
}

__global__ void field_sqrt(ptype* input, ptype* output, unsigned int N)
{
	//compute the index for this thread
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ThreadIndex1D();
	if(i >= N) return;

	output[i] = sqrt(input[i]);

}


__global__ void field_scale(bsComplex* x, bsComplex* y, bsComplex* z, unsigned int N, ptype v)
{
    //compute the index for this thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= N) return;

	if(x != NULL)
        x[i] *= v;
    if(y != NULL)
        y[i] *= v;
    if(z != NULL)
        z[i] *= v;
}


scalarslice fieldslice::Mag()
{
	//compute the magnitude of the field at each rtsPoint in the slice

    scalarslice* result = new scalarslice(R[0], R[1]);

	//compute the total number of values in the slice
	unsigned int N = R[0] * R[1];
	//int gridDim = (N+BLOCK-1)/BLOCK;
	dim3 gridDim = GenGrid1D(N, BLOCK);

	field_intensity<<<gridDim, BLOCK>>>(x_hat, y_hat, z_hat, result->S, N);
	field_sqrt<<<gridDim, BLOCK>>>(result->S, result->S, N);

	return *result;
}

scalarslice fieldslice::Real()
{
	//compute the magnitude of the field at each rtsPoint in the slice

	//create a scalar slice at the same resolution as the field
	scalarslice* result = new scalarslice(R[0], R[1]);

	//compute the total number of values in the slice
	unsigned int N = R[0] * R[1];
	//int gridDim = (N+BLOCK-1)/BLOCK;
	dim3 gridDim = GenGrid1D(N, BLOCK);

	field_real<<<gridDim, BLOCK>>>(x_hat, result->S, N);

	return *result;
}

scalarslice fieldslice::Imag()
{
	//compute the magnitude of the field at each rtsPoint in the slice

	//create a scalar slice at the same resolution as the field
	scalarslice* result = new scalarslice(R[0], R[1]);

	//compute the total number of values in the slice
	unsigned int N = R[0] * R[1];
	//int gridDim = (N+BLOCK-1)/BLOCK;
	dim3 gridDim = GenGrid1D(N, BLOCK);

	field_imaginary<<<gridDim, BLOCK>>>(x_hat, result->S, N);

	return *result;
}

void fieldslice::IntegrateAndResample(scalarslice* detector, unsigned int supersample, float scale)
{
    //compute the intensity and resample at the detector resolution
	unsigned int D[2];
	D[0] = detector->R[0];
	D[1] = detector->R[1];

	//create one thread for each detector pixel
	dim3 dimBlock(SQRT_BLOCK, SQRT_BLOCK);
	dim3 dimGrid((D[0] + SQRT_BLOCK -1)/SQRT_BLOCK, (D[1] + SQRT_BLOCK - 1)/SQRT_BLOCK);

    resample_intensity<<<dimGrid, dimBlock>>>(x_hat, y_hat, z_hat, detector->S, D[0], D[1], supersample, scale);
}

void fieldslice::ResampleAndSave(scalarslice* detector, unsigned int supersample)
{
    //compute the intensity and resample at the detector resolution
    unsigned int D[2];
    D[0] = detector->R[0];
    D[1] = detector->R[1];

    //create one thread for each detector pixel
    dim3 dimBlock(SQRT_BLOCK, SQRT_BLOCK);
    dim3 dimGrid((D[0] + SQRT_BLOCK -1)/SQRT_BLOCK, (D[1] + SQRT_BLOCK - 1)/SQRT_BLOCK);

    resample_intensity_without_integrating<<<dimGrid, dimBlock>>>(x_hat, y_hat, z_hat, detector->S, D[0], D[1], supersample);
}


scalarslice fieldslice::Intensity()
{
	//compute the magnitude of the field at each rtsPoint in the slice

	//create a scalar slice at the same resolution as the field
	scalarslice* result = new scalarslice(R[0], R[1]);

	//compute the total number of values in the slice
	unsigned int N = R[0] * R[1];
	int gridDim = (N+BLOCK-1)/BLOCK;

	field_intensity<<<gridDim, BLOCK>>>(x_hat, y_hat, z_hat, result->S, N);

	return *result;
}

void fieldslice::ScaleField(ptype v)
{
    //This function scales the field by some constant value v
    //This is mostly used for the inverse FFT, which has to divide the field by R^2

    //compute the total number of values in the slice
	unsigned int N = R[0] * R[1];
	int gridDim = (N+BLOCK-1)/BLOCK;

	field_scale<<<gridDim, BLOCK>>>(x_hat, y_hat, z_hat, N, v);

}

void fieldslice::init_gpu()
{
	//if the field has no size, return
	if(R[0] == 0 || R[1] == 0)
		return;

    //free any previous memory allocations
    if(x_hat)
        HANDLE_ERROR(hipFree(x_hat));
    if(y_hat)
        HANDLE_ERROR(hipFree(y_hat));
    if(z_hat)
        HANDLE_ERROR(hipFree(z_hat));

    //allocate space on the GPU for the field slice
	HANDLE_ERROR(hipMalloc((void**)&x_hat, R[0] * R[1] * sizeof(bsComplex)));

	if(!scalarField)
	{
		HANDLE_ERROR(hipMalloc((void**)&y_hat, R[0] * R[1] * sizeof(bsComplex)));
		//HANDLE_ERROR(hipMemset(y_hat, 0, R[0] * R[1] * sizeof(bsComplex)));

		HANDLE_ERROR(hipMalloc((void**)&z_hat, R[0] * R[1] * sizeof(bsComplex)));
		//HANDLE_ERROR(hipMemset(z_hat, 0, R[0] * R[1] * sizeof(bsComplex)));
	}

	clear_gpu();
}

void fieldslice::kill_gpu()
{
    if(x_hat != NULL)
        HANDLE_ERROR(hipFree(x_hat));
    if(y_hat != NULL)
        HANDLE_ERROR(hipFree(y_hat));
    if(z_hat != NULL)
        HANDLE_ERROR(hipFree(z_hat));

	x_hat = y_hat = z_hat = NULL;

}

void fieldslice::clear_gpu()
{
	int memsize = R[0] * R[1] * sizeof(bsComplex);
	if(x_hat != NULL)
		HANDLE_ERROR(hipMemset(x_hat, 0, memsize));
	if(y_hat != NULL)
		HANDLE_ERROR(hipMemset(y_hat, 0, memsize));
	if(z_hat != NULL)
		HANDLE_ERROR(hipMemset(z_hat, 0, memsize));

}

__global__ void copy_crop(bsComplex* source, bsComplex* dest, int u, int v, int su, int sv, int uR, int vR)
{
    //get the current coordinate in the plane slice
	int iu = blockIdx.x * blockDim.x + threadIdx.x;
	int iv = blockIdx.y * blockDim.y + threadIdx.y;

	//make sure that the thread indices are in-bounds
	if(iu >= su || iv >= sv) return;

	//compute the destination index
	int i = iv*su + iu;

	//compute the source index
	int sourceV = v + iv;
	int sourceU = u + iu;
	int is = sourceV * uR + sourceU;

	dest[i] = source[is];

}

fieldslice fieldslice::crop(int u, int v, int su, int sv)
{
	//create a new field slice with the appropriate settings
	fieldslice result(su, sv);
	result.scalarField = scalarField;

	//allocate space for the new field
	//result.init_gpu();

	//create one thread for each pixel of the field slice
	dim3 dimBlock(SQRT_BLOCK, SQRT_BLOCK);
	dim3 dimGrid((su + SQRT_BLOCK -1)/SQRT_BLOCK, (sv + SQRT_BLOCK - 1)/SQRT_BLOCK);

	//call a kernel to copy the cropped to the new field slice
	if(x_hat != NULL)
		copy_crop<<<dimGrid, dimBlock>>>(x_hat, result.x_hat, u, v, su, sv, R[0], R[1]);
	if(y_hat != NULL)
		copy_crop<<<dimGrid, dimBlock>>>(y_hat, result.y_hat, u, v, su, sv, R[0], R[1]);
	if(z_hat != NULL)
		copy_crop<<<dimGrid, dimBlock>>>(z_hat, result.z_hat, u, v, su, sv, R[0], R[1]);

	return result;
}

fieldslice::fieldslice(const fieldslice& rhs)
{
	R[0] = rhs.R[0];
	R[1] = rhs.R[1];
	scalarField = rhs.scalarField;

	x_hat = y_hat = z_hat = NULL;

	unsigned int bytes = sizeof(bsComplex) * R[0] * R[1];
	if(rhs.x_hat != NULL)
	{
		HANDLE_ERROR(hipMalloc( (void**)&x_hat, bytes));
		HANDLE_ERROR(hipMemcpy( x_hat, rhs.x_hat, bytes, hipMemcpyDeviceToDevice));
	}
	if(rhs.y_hat != NULL)
	{
		HANDLE_ERROR(hipMalloc( (void**)&y_hat, bytes));
		HANDLE_ERROR(hipMemcpy( y_hat, rhs.y_hat, bytes, hipMemcpyDeviceToDevice));
	}
	if(rhs.z_hat != NULL)
	{
		HANDLE_ERROR(hipMalloc( (void**)&z_hat, bytes));
		HANDLE_ERROR(hipMemcpy( z_hat, rhs.z_hat, bytes, hipMemcpyDeviceToDevice));
	}

}

fieldslice& fieldslice::operator=(const fieldslice& rhs)
{
	//make sure this isn't a self-allocation
	if(this != &rhs)
	{
		//make a shallow copy
		R[0] = rhs.R[0];
		R[1] = rhs.R[1];
		scalarField = rhs.scalarField;

		//initialize to new parameters
		init_gpu();

		//make a deep copy
		unsigned int bytes = sizeof(bsComplex) * R[0] * R[1];
		if(x_hat != NULL)
			HANDLE_ERROR(hipMemcpy(x_hat, rhs.x_hat, bytes, hipMemcpyDeviceToDevice));
		if(y_hat != NULL)
			HANDLE_ERROR(hipMemcpy(y_hat, rhs.y_hat, bytes, hipMemcpyDeviceToDevice));
		if(z_hat != NULL)
			HANDLE_ERROR(hipMemcpy(z_hat, rhs.z_hat, bytes, hipMemcpyDeviceToDevice));
	}

	return *this;

}
